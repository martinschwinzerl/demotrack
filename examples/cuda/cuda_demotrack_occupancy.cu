#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
// #include <cudart.h>

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <vector>
#include <sys/time.h>

#if !defined( PARTICLE_ARGPTR_DEC )
    #define PARTICLE_ARGPTR_DEC
#endif /* !defined( PARTICLE_ARGPTR_DEC ) */

#if !defined( BE_ARGPTR_DEC )
    #define BE_ARGPTR_DEC
#endif /* !defined( BE_ARGPTR_DEC ) */

#if !defined( DISABLE_SPACECHARGE )
    #define DISABLE_SPACECHARGE 1
#endif /* !defined( DISABLE_SPACECHARGE ) */

#if !defined( DEMOTRACK_STATIC )
    #define DEMOTRACK_STATIC static
#endif /* !defined( DEMOTRACK_STATIC ) */

#if !defined( DEMOTRACK_FN )
    #define DEMOTRACK_FN __device__ __host__
#endif /* !defined( DEMOTRACK_FN ) */

#if !defined( DEMOTRACK_INLINE )
    #define DEMOTRACK_INLINE inline
#endif /* !defined( DEMOTRACK_INLINE ) */

#include "src/particle.h"
#include "src/beam_elements.h"
#include "src/space_charge.h"
#include "src/track.h"

__global__ void track_particles_until_turn_nonoptimised(
    PARTICLE_ARGPTR_DEC Particle* pset, long int const num_particles,
    BE_ARGPTR_DEC double const* beam_elements_buffer,
    long int const num_slots_in_buffer, long int const until_turn )
{
    /* Stride = ( threads/block ) * ( number of blocks ) */
    long int const STRIDE = blockDim.x * gridDim.x;
    long int idx = threadIdx.x + blockIdx.x * blockDim.x;

    for( ; idx < num_particles ; idx += STRIDE )
    {
        /* Copy particle to thread-local memory  */
        Particle p = pset[ idx ];
        Track_particle_until_turn( &p,
            beam_elements_buffer, num_slots_in_buffer, until_turn );

        /* Copy results back to global mmemory */
        pset[ idx ] = p;
    }
}

int main()
{
    /* Source: https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/ */

    using std::size_t;

    /* ********************************************************************* */
    /* Prepare particle set to track */
    long int const NUM_PARTICLES = 50 * 1024;
    double const P0_C = 1.0e9; /* Kinetic energy, [eV]  */
    double const MASS0 = 938.272081e6; /* Proton rest mass, [eV] */

    std::vector< ::Particle > pset( NUM_PARTICLES, ::Particle{} );
    ::ParticleSet_setup( pset.data(), pset.size(), P0_C, MASS0 );

    /* ********************************************************************* */
    /* Read lattice from prepared and saved dump */

    size_t const LATTICE_NUM_SLOTS = 153600 / sizeof( double );
    std::vector< double > lattice( LATTICE_NUM_SLOTS, 0.0 );

    char const PATH_TO_LATTICE[] = "./demo_lattice.bin";
    FILE* fp = std::fopen( PATH_TO_LATTICE, "rb" );

    if( fp != nullptr )
    {
        size_t const ret = std::fread( lattice.data(), sizeof( double ),
            lattice.size(), fp );

        if( ret != LATTICE_NUM_SLOTS )
        {
            std::cerr << "Unable to read lattice" << std::endl;
            return 0;
        }

        std::fclose( fp );
        fp = nullptr;
    }

    /* ******************************************************************** */
    /* Prepare device memory */

    ::Particle* particles_dev = nullptr;
    double* lattice_dev = nullptr;

    auto status = ::hipMalloc( ( void** )&particles_dev,
        sizeof( ::Particle ) * NUM_PARTICLES );
    assert( status == hipSuccess );

    status = ::hipMalloc( ( void** )&lattice_dev,
        sizeof( double ) * LATTICE_NUM_SLOTS );
    assert( status == hipSuccess );

    /* Copy particle and lattice data to device */

    status = ::hipMemcpy( lattice_dev, lattice.data(),
        LATTICE_NUM_SLOTS * sizeof( double ), ::hipMemcpyHostToDevice );

    assert( status == hipSuccess );

    status = ::hipMemcpy( particles_dev, pset.data(),
        pset.size() * sizeof( ::Particle ), ::hipMemcpyHostToDevice );

    assert( status == hipSuccess );

    /* ******************************************************************** */
    /* Estimate block size */

    int BLOCK_SIZE = 0;
    int MIN_GRID_SIZE = 0;

    status = ::hipOccupancyMaxPotentialBlockSize(
        &MIN_GRID_SIZE, /* -> minimum grid size needed for max occupancy */
        &BLOCK_SIZE, /* -> estimated optimal block size */
        track_particles_until_turn_nonoptimised, /* the kernel */
        0u, /* -> dynamic shared memory per block required [bytes] */
        0u /* -> max block size limit for the kernel; 0 == no limit */ );

    assert( status == hipSuccess );

    assert( BLOCK_SIZE > 0 );
    int const GRID_SIZE = ( NUM_PARTICLES + BLOCK_SIZE - 1 ) / BLOCK_SIZE;

    std::cout << "NUM_OF_BLOCKS     : " << GRID_SIZE << "\r\n"
              << "THREADS_PER_BLOCK : " << BLOCK_SIZE << std::endl;

    /* ******************************************************************** */
    /* Perform calculation */

    long int const TRACK_UNTIL_TURN = 10;

    struct timeval  start;
    struct timeval  stop;

    ::gettimeofday( &start, NULL );

    track_particles_until_turn_nonoptimised<<< GRID_SIZE, BLOCK_SIZE >>>(
        particles_dev, NUM_PARTICLES, lattice_dev,
            LATTICE_NUM_SLOTS, TRACK_UNTIL_TURN );

    status = ::hipDeviceSynchronize();
    assert( status == hipSuccess );

    ::gettimeofday( &stop, NULL );

    std::cout << "Elapsed time: "
              << static_cast< double >( stop.tv_usec - start.tv_usec ) / 1000000.0 +
                 static_cast< double >( stop.tv_sec - start.tv_sec )
              << " seconds" << std::endl;

    /* Calculate theoretical Occupancy */
    int MAX_ACTIVE_BLOCKS;
    status = ::hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &MAX_ACTIVE_BLOCKS, /* -> returned number of max active blocks */
        track_particles_until_turn_nonoptimised, /* -> Kernel */
        BLOCK_SIZE, /* -> threads / block for kernel */
        0u /* -> dynamic shared memory per block required [bytes] */ );

    assert( status == hipSuccess );

    int device;
    ::hipDeviceProp_t props;
    status = ::hipGetDevice( &device );
    assert( status == hipSuccess );

    status = ::hipGetDeviceProperties( &props, device );
    assert( status == hipSuccess );

    double const occupancy = ( MAX_ACTIVE_BLOCKS * BLOCK_SIZE / props.warpSize) /
                    (double)(props.maxThreadsPerMultiProcessor /
                            props.warpSize);

    std::cout << "Theoretical occupancy: " << occupancy << "\r\n"
              << "max num active blocks: " << MAX_ACTIVE_BLOCKS << "\r\n"
              << std::endl;


    /* ******************************************************************** */
    /* Fetch particle data */

    status = ::hipMemcpy( pset.data(), particles_dev,
        pset.size() * sizeof( ::Particle ), ::hipMemcpyDeviceToHost );

    assert( status == hipSuccess );
    ( void )status;

    ::hipFree( lattice_dev );
    ::hipFree( particles_dev );

    return 0;
}
