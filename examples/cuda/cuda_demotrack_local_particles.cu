#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
// #include <cudart.h>

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <vector>
#include <sys/time.h>

#if !defined( PARTICLE_ARGPTR_DEC )
    #define PARTICLE_ARGPTR_DEC
#endif /* !defined( PARTICLE_ARGPTR_DEC ) */

#if !defined( BE_ARGPTR_DEC )
    #define BE_ARGPTR_DEC
#endif /* !defined( BE_ARGPTR_DEC ) */

#if !defined( DISABLE_SPACECHARGE )
    #define DISABLE_SPACECHARGE 1
#endif /* !defined( DISABLE_SPACECHARGE ) */

#if !defined( DEMOTRACK_STATIC )
    #define DEMOTRACK_STATIC static
#endif /* !defined( DEMOTRACK_STATIC ) */

#if !defined( DEMOTRACK_FN )
    #define DEMOTRACK_FN __device__ __host__
#endif /* !defined( DEMOTRACK_FN ) */

#if !defined( DEMOTRACK_INLINE )
    #define DEMOTRACK_INLINE inline
#endif /* !defined( DEMOTRACK_INLINE ) */

#include "src/particle.h"
#include "src/beam_elements.h"
#include "src/space_charge.h"
#include "src/track.h"

__global__ void track_particles_until_turn_nonoptimised(
    PARTICLE_ARGPTR_DEC Particle* pset, long int const num_particles,
    BE_ARGPTR_DEC double const* beam_elements_buffer,
    long int const num_slots_in_buffer, long int const until_turn )
{
    /* Stride = ( threads/block ) * ( number of blocks ) */
    long int const STRIDE = blockDim.x * gridDim.x;
    long int idx = threadIdx.x + blockIdx.x * blockDim.x;

    for( ; idx < num_particles ; idx += STRIDE )
    {
        /* Copy particle to thread-local memory  */
        Particle p = pset[ idx ];
        Track_particle_until_turn( &p,
            beam_elements_buffer, num_slots_in_buffer, until_turn );

        /* Copy results back to global mmemory */
        pset[ idx ] = p;
    }
}

int main()
{
    using std::size_t;

    /* ********************************************************************* */
    /* Prepare particle set to track */
    long int const NUM_PARTICLES = 50 * 1024;
    double const P0_C = 1.0e9; /* Kinetic energy, [eV]  */
    double const MASS0 = 938.272081e6; /* Proton rest mass, [eV] */

    std::vector< ::Particle > pset( NUM_PARTICLES, ::Particle{} );
    ::ParticleSet_setup( pset.data(), pset.size(), P0_C, MASS0 );

    /* ********************************************************************* */
    /* Read lattice from prepared and saved dump */

    size_t const LATTICE_NUM_SLOTS = 153600 / sizeof( double );
    std::vector< double > lattice( LATTICE_NUM_SLOTS, 0.0 );

    char const PATH_TO_LATTICE[] = "./demo_lattice.bin";
    FILE* fp = std::fopen( PATH_TO_LATTICE, "rb" );

    if( fp != nullptr )
    {
        size_t const ret = std::fread( lattice.data(), sizeof( double ),
            lattice.size(), fp );

        if( ret != LATTICE_NUM_SLOTS )
        {
            std::cerr << "Unable to read lattice" << std::endl;
            return 0;
        }

        std::fclose( fp );
        fp = nullptr;
    }

    /* ******************************************************************** */
    /* Prepare device memory */

    ::Particle* particles_dev = nullptr;
    double* lattice_dev = nullptr;

    auto status = ::hipMalloc( ( void** )&particles_dev,
        sizeof( ::Particle ) * NUM_PARTICLES );
    assert( status == hipSuccess );

    status = ::hipMalloc( ( void** )&lattice_dev,
        sizeof( double ) * LATTICE_NUM_SLOTS );
    assert( status == hipSuccess );

    /* Copy particle and lattice data to device */

    status = ::hipMemcpy( lattice_dev, lattice.data(),
        LATTICE_NUM_SLOTS * sizeof( double ), ::hipMemcpyHostToDevice );

    assert( status == hipSuccess );

    status = ::hipMemcpy( particles_dev, pset.data(),
        pset.size() * sizeof( ::Particle ), ::hipMemcpyHostToDevice );

    assert( status == hipSuccess );

    /* ******************************************************************** */
    /* Estimate block size */

    long int const THREADS_PER_BLOCK = 512;
    long int const NUM_OF_BLOCKS = ( NUM_PARTICLES +
        THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK;

    std::cout << "NUM_OF_BLOCKS     : " << NUM_OF_BLOCKS << "\r\n"
              << "THREADS_PER_BLOCK : " << THREADS_PER_BLOCK << std::endl;

    /* ******************************************************************** */
    /* Perform calculation */

    long int const TRACK_UNTIL_TURN = 10;

    struct timeval  start;
    struct timeval  stop;

    ::gettimeofday( &start, NULL );

    track_particles_until_turn_nonoptimised<<<
        NUM_OF_BLOCKS, THREADS_PER_BLOCK >>>(
            particles_dev, NUM_PARTICLES, lattice_dev,
                LATTICE_NUM_SLOTS, TRACK_UNTIL_TURN );

    status = ::hipDeviceSynchronize();
    assert( status == hipSuccess );

    ::gettimeofday( &stop, NULL );

    std::cout << "Elapsed time: "
              << static_cast< double >( stop.tv_usec - start.tv_usec ) / 1000000.0 +
                 static_cast< double >( stop.tv_sec - start.tv_sec )
              << " seconds" << std::endl;

    /* ******************************************************************** */
    /* Fetch particle data */

    status = ::hipMemcpy( pset.data(), particles_dev,
        pset.size() * sizeof( ::Particle ), ::hipMemcpyDeviceToHost );

    assert( status == hipSuccess );
    ( void )status;

    ::hipFree( lattice_dev );
    ::hipFree( particles_dev );

    return 0;
}
